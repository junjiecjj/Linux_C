#include "hip/hip_runtime.h"
/*************************************************************************
>> File Name: MatrixMultipl.c
>> Author: chenjunjie
>> Mail: 2716705056qq.com
>> Created Time: 2019.06.07
************************************************************************/

#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define Width 4
#define Block_width 2

//device code
__global__ void MatrixMulKernel(int *d_M, int *d_N, int *d_P, int width)
{
    //计算P和M的行
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    //计算P和N的列
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Row = %d, Col = %d\n", Row, Col);
    if((Row < Width) && (Col < Width))
    {
        int Pvalue = 0;
        for(int i = 0; i < Width; ++i)
        {
            Pvalue += d_M[Row*Width + i]*d_N[i*Width + Col];
        }

        d_P[Row*Width + Col] = Pvalue;
    }
}


void showaray(int *A, int row, int col)
{
    for(int i = 0; i< row; ++i)
    {
        for(int j = 0; j < col; ++j)
        {
            printf("%3d",A[i*Width + j]);
        }
        printf("\n");
    }
}
//Host code
int main(int argc, char *argv[])
{

    int dev_count;
    hipDeviceProp_t dev_prop;

    hipGetDeviceCount(&dev_count);
    printf("设备数:%d\n",dev_count);
    for(int i = 0; i < dev_count; ++i)
    {
        hipGetDeviceProperties(&dev_prop,i);
        printf("每个block支持的最大线程数: %d\n", dev_prop.maxThreadsPerBlock);
        printf("设备中SM数: %d\n", dev_prop.multiProcessorCount);
        printf("时钟频率为: %d\n",dev_prop.clockRate);
        printf("每个Block的x方向的最大线程数: %d\n", dev_prop.maxThreadsDim[0]);
        printf("每个Block的y方向的最大线程数: %d\n", dev_prop.maxThreadsDim[1]);
        printf("每个Block的z方向的最大线程数: %d\n", dev_prop.maxThreadsDim[2]);
        printf("每个grid的x方向支持的最大的block数: %d\n", dev_prop.maxGridSize[0]);
        printf("每个grid的y方向支持的最大的block数: %d\n", dev_prop.maxGridSize[1]);
        printf("每个grid的z方向支持的最大的block数: %d\n", dev_prop.maxGridSize[2]);
        printf("每个warp包含的线程数为: %d\n", dev_prop.warpSize);
        printf("每个SM里面的registers变量个数:%d\n", dev_prop.regsPerBlock);
        printf("每个Block里共享内存:%lu\n", dev_prop.sharedMemPerBlock);
    }

    int N = Width * Width;
    size_t size = Width * Width * sizeof(int);

    int *h_M = (int *)malloc(size);
    int *h_N = (int *)malloc(size);
    int *h_P = (int *)malloc(size);
    //初始化h_M, h_N;
    for(int i = 0; i < N; ++i)
    {
        h_M[i] = 1;
        h_N[i] = 1;
        h_P[i] = 0;
    }
    printf("h_M, h_N,h_P分配成功\n");
    printf("打印h_M\n");
    showaray(h_M, Width, Width);
    printf("打印h_N\n");
    showaray(h_N, Width, Width);
    printf("打印h_P\n");
    showaray(h_P, Width, Width);

    int *d_M;
    hipMalloc(&d_M, size);

    int *d_N;
    hipMalloc(&d_N, size);

    int *d_P;
    hipMalloc(&d_P, size);
    printf("d_M, d_N, d_P分配成功\n");

    hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);
    printf("d_M, d_N, d_P拷贝到h成功\n");


    int NumBlocks = Width/Block_width;
    if(Width%Block_width )
        NumBlocks++;

    dim3 grid(NumBlocks, NumBlocks, 1);
    dim3 block(Block_width, Block_width, 1);
    MatrixMulKernel<<<grid, block>>>(d_M, d_N, d_P, Width);

    hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);
    printf("计算成功\n");

    showaray(h_P, Width, Width);
    hipFree(d_M);
    hipFree(d_M);
    hipFree(d_P);
    printf("释放d_M, d_P, d_N成功\n");
    return 0;
}
